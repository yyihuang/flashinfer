/*
 * Copyright (c) 2024 by FlashInfer team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hip/driver_types.h>

#include <flashinfer/gemm/bmm_fp8.cuh>

#include "pytorch_extension_utils.h"

void bmm_fp8(at::Tensor A, at::Tensor B, at::Tensor D, at::Tensor A_scale, at::Tensor B_scale,
             at::Tensor workspace_buffer, int64_t cublas_handle, int64_t hip_stream) {
  TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
  TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
  TORCH_CHECK(D.is_cuda(), "D must be a CUDA tensor");
  TORCH_CHECK(A.dim() == 3, "Expected 3D tensor for A");
  TORCH_CHECK(B.dim() == 3, "Expected 3D tensor for B");
  TORCH_CHECK(D.dim() == 3, "Expected 3D tensor for D");
  TORCH_CHECK(A.size(0) == B.size(0) && A.size(0) == D.size(0), "Batch sizes must match");
  TORCH_CHECK(A.size(2) == B.size(1), "Incompatible matrix sizes");
  TORCH_CHECK(A.size(1) == D.size(1) && B.size(2) == D.size(2),
              "Result tensor has incorrect shape");

  // PyTorch is row major by default. cuBLASLt is column major by default.
  // We need row major D as expected.
  // A ^ T * B = D, so D ^ T = B ^ T * A
  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(B.scalar_type(), b_type, [&] {
    return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(A.scalar_type(), a_type, [&] {
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(D.scalar_type(), d_type, [&] {
        auto batch_size = A.size(0);
        auto m = A.size(1);
        auto k = A.size(2);
        auto n = B.size(2);

        auto lt_handle = reinterpret_cast<hipblasLtHandle_t>(cublas_handle);

        auto stream = reinterpret_cast<hipStream_t>(hip_stream);

        int smCount = 32;
        /* Get CUDA stream device */
        hipError_t cu_result;

        int device;
        hipDeviceGet(&device, 0);
        // auto cuda_err = hipStreamGetAttribute(stream, cudaStreamAttributeDevice, &device);
        // auto cuda_err = cudaStreamGetDevice(stream, &device);
        // TORCH_CHECK(cuda_err == hipSuccess, "cudaStreamGetDevice failed: ", cublasGetStatusString(cuda_err));

        CUdevResource resource_all;
        cu_result = cuDeviceGetDevResource(device, &resource_all, CU_DEV_RESOURCE_TYPE_SM);
        TORCH_CHECK(cu_result == hipSuccess, "cuDeviceGetDevResource failed");
        CUdevResource resource_split;
        unsigned int one = 1;
        cu_result = cuDevSmResourceSplitByCount(&resource_split, &one, &resource_all, NULL, 0, smCount);
        TORCH_CHECK(cu_result == hipSuccess, "cuDevSmResourceSplitByCount failed");
        CUdevResourceDesc resourceDesc;
        cu_result = cuDevResourceGenerateDesc(&resourceDesc, &resource_split, 1);
        TORCH_CHECK(cu_result == hipSuccess, "cuDevResourceGenerateDesc failed");

        /* Create CUDA green context stream */
        CUgreenCtx green_ctx;
        cu_result = cuGreenCtxCreate(&green_ctx, resourceDesc, device, CU_GREEN_CTX_DEFAULT_STREAM);
        TORCH_CHECK(cu_result == hipSuccess, "cuGreenCtxCreate failed");
        hipStream_t new_stream;
        cu_result = cuGreenCtxStreamCreate(&new_stream, green_ctx, hipStreamNonBlocking, 0);
        TORCH_CHECK(cu_result == hipSuccess, "cuGreenCtxStreamCreate failed");

        hipCtx_t context;
        cu_result = cuCtxFromGreenCtx(&context, green_ctx);
        TORCH_CHECK(cu_result == hipSuccess, "cuCtxFromGreenCtx failed");
        cu_result = hipCtxPushCurrent(context);
        TORCH_CHECK(cu_result == hipSuccess, "hipCtxPushCurrent failed");

        auto status = flashinfer::bmm_fp8::bmm_fp8_internal_cublaslt(
            workspace_buffer.data_ptr(), workspace_buffer.numel(),
            static_cast<b_type*>(B.data_ptr()), static_cast<a_type*>(A.data_ptr()),
            static_cast<d_type*>(D.data_ptr()), batch_size, n, m, k,
            static_cast<float*>(B_scale.data_ptr()), static_cast<float*>(A_scale.data_ptr()),
            lt_handle, new_stream);
        TORCH_CHECK(status == HIPBLAS_STATUS_SUCCESS,
                    "bmm_fp8_internal_cublaslt failed: ", cublasGetStatusString(status));

        cu_result = hipCtxPopCurrent(NULL);
        TORCH_CHECK(cu_result == hipSuccess, "hipCtxPopCurrent failed");
        cu_result = cuGreenCtxDestroy(green_ctx);
        TORCH_CHECK(cu_result == hipSuccess, "cuGreenCtxDestroy failed");

        return true;
      });
    });
  });
}
